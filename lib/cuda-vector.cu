#include "hip/hip_runtime.h"
#ifndef COMMON_H
#include "../src/common.h"
#endif 

__global__ void sum(int* first, int* second, int* result, int width){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < width)
    result[idx] = first[idx] + second[idx];
  
  

}

struct Vector* VectorSum(struct Vector* first, struct Vector* second){

  if(first->position != 1 || second->position != 1 ){
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(first->width/10+1);
    dim3 block_size(first->width/10 + 1 + (first->width%10 == 0 ? 0 : 1));
    sum <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return NULL;
}      
