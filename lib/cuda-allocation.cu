#ifndef COMMON_H
#include "../include/common.h"
#endif 
// MatrixAllocateOnDevice allocate an amount of memory on the device and set the pitch and the width and the height of the matrix

struct Matrix* MatrixAllocateOnDevice(int width, int height){
  
  struct Matrix *d_tmp;
  d_tmp = (struct Matrix*) malloc(sizeof(struct Matrix));
  d_tmp->x = width;
  d_tmp->y = height;
  d_tmp->position = 1;
  hipMalloc((void**)&d_tmp->matrix, sizeof(int)*width*height);
  return d_tmp;
}

// MatrixAllocateOnHost allocate an amount of memory on host and set width, height,  pitch and allocate matrix

struct Matrix* MatrixAllocateOnHost(int width, int height){

  struct Matrix *h_tmp;
  h_tmp = (struct Matrix*) malloc(sizeof(struct Matrix));
  h_tmp->matrix = (int*) malloc ( width * height * sizeof(int));
  h_tmp->x = width;
  h_tmp->y = height;
  h_tmp->position = 0;
  return h_tmp;

}

struct Matrix* GetMatrixFromDevice(struct Matrix* d_m){

  struct Matrix* h_m = MatrixAllocateOnHost(d_m->x, d_m->y);
  hipMemcpy(h_m->matrix, d_m->matrix, sizeof(int)*d_m->x*d_m->y, hipMemcpyDeviceToHost);
  
  return h_m;

}

struct Matrix* SetMatrixOnDevice(struct Matrix* h_m){
  
  struct Matrix* d_m = MatrixAllocateOnDevice(h_m->x,h_m->y);
  hipMemcpy(d_m->matrix, h_m->matrix, sizeof(int)*h_m->x*h_m->y,hipMemcpyHostToDevice);
  return d_m;
  
}

struct Vector* VectorAllocateOnDevice(int width){

  struct Vector *d_m;

  d_m = (struct Vector*) malloc(sizeof(struct Vector));
  d_m->width = width;
  d_m->vector = (int*) malloc(sizeof(int) * width);
  d_m->position = 1;
  memset(d_m->vector,0, sizeof(int) * width);
  hipMalloc((void**)&d_m->vector, sizeof(int) * width);

  return d_m;
  

}

struct Vector* VectorAllocateOnHost(int width){


  struct Vector* h_m;
  h_m = (struct Vector*) malloc(sizeof(struct Vector));
  h_m->vector = (int*) malloc(sizeof(int) * width);
  h_m->width = width;
  memset(h_m->vector,0, sizeof(int) * width);
  h_m->position = 0;
  return h_m;

}

struct Vector* GetVectorFromDevice(struct Vector* d_m){


  struct Vector* h_m = VectorAllocateOnHost(d_m->width);
  hipMemcpy(h_m->vector, d_m->vector, sizeof(int) * d_m->width, hipMemcpyDeviceToHost);
  return h_m;
    
};


struct Vector* SetVectorOnDevice(struct Vector* h_m){

  struct Vector* d_m = VectorAllocateOnDevice(h_m->width);
  hipMemcpy(d_m->vector, h_m->vector, sizeof(int) * h_m->width, hipMemcpyHostToDevice);
  return d_m;
}

