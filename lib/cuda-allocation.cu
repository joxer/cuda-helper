#include "../src/common.h"

/*

====


 Matrix Part 


====


*/




// MatrixAllocateOnDevice allocate an amount of memory on the device and set the pitch and the width and the height of the matrix


struct Matrix* MatrixAllocateOnDevice(int width, int height){
  
  struct Matrix *d_tmp;
  d_tmp = (struct Matrix*) malloc(sizeof(struct Matrix));
  d_tmp->x = width;
  d_tmp->y = height;

  hipMallocPitch((void**)&(d_tmp->matrix), &(d_tmp->pitch), width * sizeof(int), height);
  return d_tmp;
}

// MatrixAllocateOnHost allocate an amount of memory on host and set width, height,  pitch and allocate matrix

struct Matrix* MatrixAllocateOnHost(int width, int height){


  struct Matrix *h_tmp;
  h_tmp = (struct Matrix*) malloc(sizeof(struct Matrix));
  h_tmp->matrix = (int*) malloc ( width * height * sizeof(int));
  h_tmp->x = width;
  h_tmp->y = height;
  h_tmp->pitch = h_tmp->x * sizeof(int);
  return h_tmp;

}

struct Matrix* GetMatrixFromDevice(struct Matrix* d_m){

  struct Matrix* h_m;
  h_m = (struct Matrix *) malloc (sizeof(struct Matrix));
  h_m->x = d_m->x;
  h_m->y = d_m->y;
  h_m->pitch = h_m->x * sizeof(int);
  h_m->matrix = (int*) malloc(sizeof(int) * h_m->x * h_m->y); // i'm faggot; by joxer
  hipMemcpy2D(h_m->matrix, h_m->pitch, d_m->matrix, d_m->pitch, h_m->x, h_m->y,hipMemcpyDeviceToHost);
  
  return h_m;

}

struct Matrix* SetMatrixOnDevice(struct Matrix* h_m){
  
  struct Matrix* d_m = MatrixAllocateOnDevice(h_m->x,h_m->y);
  hipMemcpy2D(d_m->matrix, d_m->pitch, h_m->matrix, h_m->x*sizeof(int), h_m->x * sizeof(int) , h_m->y,hipMemcpyDeviceToHost);
  return d_m;
  
}

/*
  
===

End Matrix Part

===
  
  
 */


/*

===

  Begin Vector Part

===  
  
 */



struct Vector* VectorAllocateOnDevice(int width){

  struct Vector *d_m;

  d_m = (struct Vector*) malloc(sizeof(struct Vector));
  d_m->width = width;
  d_m->vector = (int*) malloc(sizeof(int) * width);
  memset(d_m->vector,0, sizeof(int) * width);
  hipMalloc((void**)&d_m->vector, sizeof(int) * width);
  return d_m;
  

}

struct Vector* VectorAllocateOnHost(int width){


  struct Vector* h_m;
  h_m = (struct Vector*) malloc(sizeof(struct Vector));
  h_m->vector = (int*) malloc(sizeof(int) * width);
  h_m->width = width;
  memset(h_m->vector,0, sizeof(int) * width);
  
  return h_m;

}

struct Vector* GetVectorFromDevice(struct Vector* d_m){


  struct Vector* h_m;
  h_m = (struct Vector*) malloc(sizeof(struct Vector));
  h_m->width = d_m->width;
  h_m->vector = (int*) malloc(sizeof(int) * h_m->width);
  memset(h_m->vector, 0 , sizeof(int) * h_m->width);
  hipMemcpy(h_m->vector, d_m->vector, sizeof(int) * d_m->width, hipMemcpyDeviceToHost);
  return h_m;
    
};


struct Vector* SetVectorOnDevice(struct Vector* h_m){

  struct Vector* d_m = VectorAllocateOnDevice(h_m->width);
  hipMemcpy(d_m->vector, h_m->vector, sizeof(int) * h_m->width, hipMemcpyHostToDevice);
  return d_m;
}
