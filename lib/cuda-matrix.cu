#include "hip/hip_runtime.h"
#include <stdio.h>
#ifndef COMMON_H
#include "../src/common.h"
#endif

__global__ void sum(int *a, int* b , int* c, size_t pitch, int xx, int yy){

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x < xx && y < yy){
    int *arow = (int *) ((char*)a + x * pitch);
    int *brow = (int *) ((char*)b + x * pitch);
    int *crow = (int *) ((char*)c + x * pitch);
    crow[y] = brow[y] + arow[y];
  }
}

struct Matrix* MatrixSum(struct Matrix* first, struct Matrix* second){
  
  int n = first->x * first->y / 512 * 512 + (first->x * first->y)%(512*512);
  
  struct Matrix* tmp_d = MatrixAllocateOnDevice(first->x, first->y);
  
  dim3 block((first->x * first->y )/ (512*512) + 1);
  dim3 block_s(512,512);
  sum <<<block, block_s>>> ( first->matrix, second->matrix, tmp_d->matrix, first->pitch, first->x, first->y);
  
  struct Matrix* tmp_h = GetMatrixFromDevice(tmp_d);
  return tmp_h;
  
  }
/*

struct Matrix* MatrixSub(struct Matrix* first, struct Matrix* second);
struct Matrix* MatrixMul(struct Matrix* first, struct Matrix* second);
struct Matrix* MatrixDiv(struct Matrix* first, struct Matrix* second);


*/
