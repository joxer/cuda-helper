#include "hip/hip_runtime.h"
#include <stdio.h>
#ifndef COMMON_H
#include "../src/common.h"
#endif

__global__ void sum(int *a, int* b , int* c, size_t pitch){

  int x = threadIdx.x;
  int y = threadIdx.y;
  
  int *arow = (int *) ((char*)a + x * pitch);
  int *brow = (int *) ((char*)b + x * pitch);
  int *crow = (int *) ((char*)c + x * pitch);
  crow[y] = brow[y] + arow[y];

}

struct Matrix* MatrixSum(struct Matrix* first, struct Matrix* second){
  
  
  //printf(" %d ", first->pitch);
    struct Matrix* tmp_d = MatrixAllocateOnDevice(first->x, first->y);
    
    dim3 block(1);
    dim3 block_s(4,4);
    sum <<<block, block_s>>> ( first->matrix, second->matrix, tmp_d->matrix, first->pitch);
    
    struct Matrix* tmp_h = GetMatrixFromDevice(tmp_d);
    return tmp_h;
  
  }
/*

struct Matrix* MatrixSub(struct Matrix* first, struct Matrix* second);
struct Matrix* MatrixMul(struct Matrix* first, struct Matrix* second);
struct Matrix* MatrixDiv(struct Matrix* first, struct Matrix* second);


*/
