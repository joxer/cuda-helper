#include "hip/hip_runtime.h"
#ifndef COMMON_H
#include "../src/common.h"
#endif 

__global__ void sum(int* first, int* second, int* result, int width){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < width)
    result[idx] = first[idx] + second[idx];
  
  

}

__global__ void sub(int* first, int* second, int* result, int width){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < width)
    result[idx] = first[idx] - second[idx];
  
}

__global__ void mul(int* first, int* second, int* result, int width){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < width)
    result[idx] = first[idx] * second[idx];
  
}


struct Vector* VectorSum(struct Vector* first, struct Vector* second){
  int n = first->width/512 + 512;
  if(first->position != 1 || second->position != 1 ){
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(n);
    dim3 block_size(512);
    sum <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return NULL;
}      

struct Vector* VectorSub(struct Vector* first, struct Vector* second){

  if(first->position != 1 || second->position != 1 ){
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(first->width/10+1);
    dim3 block_size(first->width/10 + 1 + (first->width%10 == 0 ? 0 : 1));
    sub <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return NULL;
}      

struct Vector* VectorMul(struct Vector* first, struct Vector* second){

  if(first->position != 1 || second->position != 1 ){
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(first->width/10+1);
    dim3 block_size(first->width/10 + 1 + (first->width%10 == 0 ? 0 : 1));
    mul <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return NULL;
}      

