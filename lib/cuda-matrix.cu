#include "hip/hip_runtime.h"
#include <stdio.h>
#ifndef COMMON_H
#include "../src/common.h"
#endif

__global__ void sum(int *a, int* b , int* c, size_t pitch, int xx, int yy){

   __shared__ int Pp[xx][yy];
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int *arow = (int *) ((char*)a + x * pitch);
  int *brow = (int *) ((char*)b + x * pitch);
  int *crow = (int *) ((char*)c + x * pitch);
   
  Pp[x][y] = arow[y] + brow[y];
  //  crow[y] = brow[y] + arow[y];
  crow[y] = Pp[x][y];
}


struct Matrix* MatrixSum(struct Matrix* first, struct Matrix* second){
  
  int n = (first->x * first->y )/ (512 * 512) + 512;
  
  struct Matrix* tmp_d = MatrixAllocateOnDevice(first->x, first->y);
  

  dim3 block(1);
  dim3 block_s(5,5);
  sum <<<block, block_s>>> ( first->matrix, second->matrix, tmp_d->matrix, first->pitch, first->x, first->y);
  
  struct Matrix* tmp_h = GetMatrixFromDevice(tmp_d);
  return tmp_h;
  
  }
/*

struct Matrix* MatrixSub(struct Matrix* first, struct Matrix* second);
struct Matrix* MatrixMul(struct Matrix* first, struct Matrix* second);
struct Matrix* MatrixDiv(struct Matrix* first, struct Matrix* second);


*/
