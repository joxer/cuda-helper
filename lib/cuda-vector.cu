#include "hip/hip_runtime.h"
#ifndef CUDA_ALLOCATION_H
#include <cuda-helper/cuda-allocation.h>
#endif
__global__ void sum(struct Vector* first, struct Vector* second, struct Vector* result, int width){

  int idx = ThreadIdx.x;
  
  if(idx < N)
    result[idx] = first[idx] + second[idx];
  
  

}

struct Vector* VectorSum(struct Vector* first, struct Vector* second){


  if((first->position != 1 || second->position != 1 ) && first->width == second->width){
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(256);
    dim3 block_size(first->width/256 + (first->width%256 == 0 ? 0 : 1));
    sum <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return null;
}      
