#ifndef COMMON_H
#include "../include/common.h"
#endif

int HostToDevice(struct Vector* first, struct Vector* second){


  if(first->position != second->position){

    hipMemcpy(second->vector, first->vector, sizeof(int)*first->width, hipMemcpyHostToDevice);
    return 1;
  }
  return 0;

}

int DeviceToHost(struct Vector* first, struct Vector* second){


  if(first->position != second->position){

    hipMemcpy(second->vector, first->vector, sizeof(int)*first->width, hipMemcpyDeviceToHost);
    return 1;
  }
  return 0;

}
