#include "hip/hip_runtime.h"
#ifndef COMMON_H
#include "../src/common.h"
#endif 

__global__ void sum(int* first, int* second, int* result, int width){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < width)
    result[idx] = first[idx] + second[idx];
  
  

}

__global__ void sub(int* first, int* second, int* result, int width){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < width)
    result[idx] = first[idx] - second[idx];
  
}

__global__ void mul(int* first, int* second, int* result, int width){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < width)
    result[idx] = first[idx] * second[idx];
  
}

__global__ void div(int* first, int* second, int* result, int width){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if(idx < width)
    result[idx] = first[idx] / second[idx];
  
}




struct Vector* VectorSum(struct Vector* first, struct Vector* second){
  
  if(first->position != 1 || second->position != 1 ){
    int n = first->width/512 + first->width%512;
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(n);
    dim3 block_size(512);
    sum <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return NULL;
}      

struct Vector* VectorSub(struct Vector* first, struct Vector* second){
  
  if(first->position != 1 || second->position != 1 ){
    int n = first->width/512 + first->width%512;
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(n);
    dim3 block_size(512);
    sub <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return NULL;
}      

struct Vector* VectorMul(struct Vector* first, struct Vector* second){

  if(first->position != 1 || second->position != 1 ){
    int n = first->width/512 + first->width%512;
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(n);
    dim3 block_size(512);
    mul <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return NULL;
}      

struct Vector* VectorDiv(struct Vector* first, struct Vector* second){

  if(first->position != 1 || second->position != 1 ){
    int n = first->width/512 + first->width%512;
    struct Vector* tmp_d = VectorAllocateOnDevice(first->width);
    dim3 dimblock(n);
    dim3 block_size(512);
    div <<<dimblock, block_size>>> (first->vector, second->vector, tmp_d->vector, first->width);
    struct Vector* tmp_h = GetVectorFromDevice(tmp_d);
    return tmp_h;
  }
  return NULL;
}      

